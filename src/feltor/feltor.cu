#include "hip/hip_runtime.h"
#include <iostream>
#include <iomanip>
#include <vector>
#include <sstream>
#include <cmath>
// #define DG_DEBUG

#include "draw/host_window.h"
//#include "draw/device_window.cuh"
#include "dg/backend/xspacelib.cuh"
#include "dg/backend/timer.cuh"
#include "file/read_input.h"
#include "solovev/geometry.h"

#include "feltor.cuh"
#include "parameters.h"

/*
   - reads parameters from input.txt or any other given file, 
   - integrates the Feltor - functor and 
   - directly visualizes results on the screen using parameters in window_params.txt
*/


int main( int argc, char* argv[])
{
    ////////////////////////Parameter initialisation//////////////////////////
    std::vector<double> v,v2,v3;
    std::stringstream title;
    if( argc == 1)
    {
        try{
            v = file::read_input("input.txt");
            v3 = file::read_input( "geometry_params.txt"); 
        }catch( toefl::Message& m){
            m.display();
            return -1;
        }
    }
    else if( argc == 3)
    {
        try{
            v = file::read_input(argv[1]);
            v3 = file::read_input( argv[2]); 
        }catch( toefl::Message& m){
            m.display();
            return -1;
        }
    }
    else
    {
        std::cerr << "ERROR: Wrong number of arguments!\nUsage: "<< argv[0]<<" [inputfile] [geomfile] \n";
        return -1;
    }
    const eule::Parameters p( v);
    p.display( std::cout);
    const solovev::GeomParameters gp(v3);
    gp.display( std::cout);
    v2 = file::read_input( "window_params.txt");
    GLFWwindow* w = draw::glfwInitAndCreateWindow( (p.Nz+1)/v2[2]*v2[3], v2[1]*v2[4], "");
    draw::RenderHostData render(v2[1], (p.Nz+1)/v2[2]);



    //////////////////////////////////////////////////////////////////////////
    double Rmin=gp.R_0-p.boxscale*gp.a;
    double Zmin=-p.boxscale*gp.a*gp.elongation;
    double Rmax=gp.R_0+p.boxscale*gp.a; 
    double Zmax=p.boxscale*gp.a*gp.elongation;
    //Make grid
     dg::Grid3d<double > grid( Rmin,Rmax, Zmin,Zmax, 0, 2.*M_PI, p.n, p.Nx, p.Ny, p.Nz, dg::DIR, dg::DIR, dg::PER, dg::cylindrical);  
    //create RHS 
    std::cout << "Constructing Feltor...\n";
    eule::Feltor<dg::DMatrix, dg::DVec, dg::DVec > feltor( grid, p,gp); //initialize before rolkar!
    std::cout << "Constructing Rolkar...\n";
    eule::Rolkar<dg::DMatrix, dg::DVec, dg::DVec > rolkar( grid, p,gp);
    std::cout << "Done!\n";

    /////////////////////The initial field///////////////////////////////////////////
    //initial perturbation
//     dg::Gaussian3d init0(gp.R_0+p.posX*gp.a, p.posY*gp.a, M_PI, p.sigma, p.sigma, p.sigma, p.amp);
//     dg::Gaussian init0( gp.R_0+p.posX*gp.a, p.posY*gp.a, p.sigma, p.sigma, p.amp);
    dg::BathRZ init0(16,16,p.Nz,Rmin,Zmin, 30.,5.,p.amp);
//     solovev::ZonalFlow init0(p, gp);
//     dg::CONSTANT init0( 0.);

    
    //background profile
    solovev::Nprofile prof(p, gp); //initial background profile
    std::vector<dg::DVec> y0(4, dg::evaluate( prof, grid)), y1(y0); 
    
    //field aligning
//     dg::CONSTANT gaussianZ( 1.);
    dg::GaussianZ gaussianZ( M_PI, p.sigma_z*M_PI, 1);
    y1[1] = feltor.dz().evaluate( init0, gaussianZ, (unsigned)p.Nz/2, 3); //rounds =2 ->2*2-1
    y1[2] = dg::evaluate( gaussianZ, grid);
    dg::blas1::pointwiseDot( y1[1], y1[2], y1[1]);
    //no field aligning
//     y1[1] = dg::evaluate( init0, grid);
    
    dg::blas1::axpby( 1., y1[1], 1., y0[1]); //initialize ni
    dg::blas1::transform(y0[1], y0[1], dg::PLUS<>(-1)); //initialize ni-1
    dg::blas1::pointwiseDot(rolkar.damping(),y0[1], y0[1]); //damp with gaussprofdamp
    std::cout << "intiialize ne" << std::endl;
    feltor.initializene( y0[1], y0[0]);    
    std::cout << "Done!\n";

    dg::blas1::axpby( 0., y0[2], 0., y0[2]); //set Ue = 0
    dg::blas1::axpby( 0., y0[3], 0., y0[3]); //set Ui = 0

    dg::Karniadakis< std::vector<dg::DVec> > karniadakis( y0, y0[0].size(), p.eps_time);
    std::cout << "intiialize karniadakis" << std::endl;
    karniadakis.init( feltor, rolkar, y0, p.dt);
    std::cout << "Done!\n";
    std::cout << "first karniadakis" << std::endl;

    karniadakis( feltor, rolkar, y0); //now energies and potential are at time 0
    std::cout << "Done!\n";

    dg::DVec dvisual( grid.size(), 0.);
    dg::HVec hvisual( grid.size(), 0.), visual(hvisual),avisual(hvisual);
    dg::HMatrix equi = dg::create::backscatter( grid);
    draw::ColorMapRedBlueExtMinMax colors(-1.0, 1.0);

    //create timer
    dg::Timer t;
    double time = 0;
    unsigned step = 0;
    
    const double mass0 = feltor.mass(), mass_blob0 = mass0 - grid.lx()*grid.ly();
    double E0 = feltor.energy(), energy0 = E0, E1 = 0., diff = 0.;
    std::cout << "Begin computation \n";
    std::cout << std::scientific << std::setprecision( 2);
    
    while ( !glfwWindowShouldClose( w ))
    {

        hvisual = karniadakis.last()[0];
        dg::blas2::gemv( equi, hvisual, visual);
        colors.scalemax() = (float)thrust::reduce( visual.begin(), visual.end(), 0., thrust::maximum<double>() );
        colors.scalemin() = -colors.scalemax();        
        //colors.scalemin() = 1.0;
        //colors.scalemin() =  (float)thrust::reduce( visual.begin(), visual.end(), colors.scalemax()  ,thrust::minimum<double>() );

        title << std::setprecision(2) << std::scientific;
        //title <<"ne / "<<(float)thrust::reduce( visual.begin(), visual.end(), colors.scalemax()  ,thrust::minimum<double>() )<<"  " << colors.scalemax()<<"\t";
        title <<"ne-1 / " << colors.scalemax()<<"\t";
        dg::blas1::axpby(0.0,avisual,0.0,avisual);
        for( unsigned k=0; k<p.Nz/v2[2];k++)
        {
            unsigned size=grid.n()*grid.n()*grid.Nx()*grid.Ny();
            dg::HVec part( visual.begin() + k*v2[2]*size, visual.begin()+(k*v2[2]+1)*size);
            dg::blas1::axpby(1.0,part,1.0,avisual);
            render.renderQuad( part, grid.n()*grid.Nx(), grid.n()*grid.Ny(), colors);
        }
        dg::blas1::scal(avisual,1./p.Nz);
        render.renderQuad( avisual, grid.n()*grid.Nx(), grid.n()*grid.Ny(), colors);
        //draw ions
        //thrust::transform( y1[1].begin(), y1[1].end(), dvisual.begin(), dg::PLUS<double>(-0.));//ne-1
        hvisual = karniadakis.last()[1];
        dg::blas2::gemv( equi, hvisual, visual);
        colors.scalemax() = (float)thrust::reduce( visual.begin(), visual.end(), 0., thrust::maximum<double>() );
        //colors.scalemin() = 1.0;        
        colors.scalemin() = -colors.scalemax();        
        //colors.scalemin() =  (float)thrust::reduce( visual.begin(), visual.end(), colors.scalemax()  ,thrust::minimum<double>() );

        title << std::setprecision(2) << std::scientific;
        //title <<"ni / "<<(float)thrust::reduce( visual.begin(), visual.end(), colors.scalemax()  ,thrust::minimum<double>() )<<"  " << colors.scalemax()<<"\t";
        title <<"ni-1 / " << colors.scalemax()<<"\t";
        dg::blas1::axpby(0.0,avisual,0.0,avisual);
        for( unsigned k=0; k<p.Nz/v2[2];k++)
        {
            unsigned size=grid.n()*grid.n()*grid.Nx()*grid.Ny();
            dg::HVec part( visual.begin() + k*v2[2]*size, visual.begin()+(k*v2[2]+1)*size);
            dg::blas1::axpby(1.0,part,1.0,avisual);
            render.renderQuad( part, grid.n()*grid.Nx(), grid.n()*grid.Ny(), colors);
        }
        dg::blas1::scal(avisual,1./p.Nz);
        render.renderQuad( avisual, grid.n()*grid.Nx(), grid.n()*grid.Ny(), colors);
        
        //draw potential
        //transform to Vor
//         dvisual=feltor.potential()[0];
//         dg::blas2::gemv( rolkar.laplacianM(), dvisual, y1[1]);
//         hvisual = y1[1];
        hvisual = feltor.potential()[0];
        dg::blas2::gemv( equi, hvisual, visual);
        colors.scalemax() = (float)thrust::reduce( visual.begin(),visual.end(), 0.,thrust::maximum<double>()  );
//         colors.scalemin() =  (float)thrust::reduce( visual.begin(), visual.end(), colors.scalemax()  ,thrust::minimum<double>() );
        colors.scalemin() = -colors.scalemax();
        //title <<"Phi / "<<colors.scalemin()<<"  " << colors.scalemax()<<"\t";
        title <<"Omega / "<< colors.scalemax()<<"\t";
        dg::blas1::axpby(0.0,avisual,0.0,avisual);
        for( unsigned k=0; k<p.Nz/v2[2];k++)
        {
            unsigned size=grid.n()*grid.n()*grid.Nx()*grid.Ny();
            dg::HVec part( visual.begin() + k*v2[2]*size, visual.begin()+(k*v2[2]+1)*size);
            dg::blas1::axpby(1.0,part,1.0,avisual);
            render.renderQuad( part, grid.n()*grid.Nx(), grid.n()*grid.Ny(), colors);
        }
        dg::blas1::scal(avisual,1./p.Nz);
        render.renderQuad( avisual, grid.n()*grid.Nx(), grid.n()*grid.Ny(), colors);

        //draw U_e
        hvisual = karniadakis.last()[2];
        dg::blas2::gemv( equi, hvisual, visual);
        colors.scalemax() = (float)thrust::reduce( visual.begin(), visual.end(), 0.,thrust::maximum<double>()  );
        //colors.scalemin() =  (float)thrust::reduce( visual.begin(), visual.end(), colors.scalemax()  ,thrust::minimum<double>() );
        colors.scalemin() = -colors.scalemax();
        //title <<"Ue / "<<colors.scalemin()<<"  " << colors.scalemax()<<"\t";
        title <<"Ue / " << colors.scalemax()<<"\t";
        dg::blas1::axpby(0.0,avisual,0.0,avisual);
        for( unsigned k=0; k<p.Nz/v2[2];k++)
        {
            unsigned size=grid.n()*grid.n()*grid.Nx()*grid.Ny();
            dg::HVec part( visual.begin() + k*v2[2]*size, visual.begin()+(k*v2[2]+1)*size);
            dg::blas1::axpby(1.0,part,1.0,avisual);
            render.renderQuad( part, grid.n()*grid.Nx(), grid.n()*grid.Ny(), colors);
        }
        dg::blas1::scal(avisual,1./p.Nz);
        render.renderQuad( avisual, grid.n()*grid.Nx(), grid.n()*grid.Ny(), colors);
        //draw U_i
        hvisual = karniadakis.last()[3];
        dg::blas2::gemv( equi, hvisual, visual);
        colors.scalemax() = (float)thrust::reduce( visual.begin(), visual.end(), 0., thrust::maximum<double>()  );
        //colors.scalemin() =  (float)thrust::reduce( visual.begin(), visual.end(), colors.scalemax()  ,thrust::minimum<double>() );
        colors.scalemin() = -colors.scalemax();
        //title <<"Ui / "<<colors.scalemin()<< "  " << colors.scalemax()<<"\t";
        title <<"Ui / " << colors.scalemax()<<"\t";
        dg::blas1::axpby(0.0,avisual,0.0,avisual);
        for( unsigned k=0; k<p.Nz/v2[2];k++)
        {
            unsigned size=grid.n()*grid.n()*grid.Nx()*grid.Ny();
            dg::HVec part( visual.begin() + k*v2[2]*size, visual.begin()+(k*v2[2]+1)*size);
            dg::blas1::axpby(1.0,part,1.0,avisual);
            render.renderQuad( part, grid.n()*grid.Nx(), grid.n()*grid.Ny(), colors);
        }
        dg::blas1::scal(avisual,1./p.Nz);
        render.renderQuad( avisual, grid.n()*grid.Nx(), grid.n()*grid.Ny(), colors);
        
        title << std::fixed; 
        title << " &&   time = "<<time;
        glfwSetWindowTitle(w,title.str().c_str());
        title.str("");
        glfwPollEvents();
        glfwSwapBuffers( w);

        //step 
#ifdef DG_BENCHMARK
        t.tic();
#endif//DG_BENCHMARK
        for( unsigned i=0; i<p.itstp; i++)
        {
            step++;
            std::cout << "(m_tot-m_0)/m_0: "<< (feltor.mass()-mass0)/mass_blob0<<"\t";
            E1 = feltor.energy();
            diff = (E1 - E0)/p.dt; //
            double diss = feltor.energy_diffusion( );
            std::cout << "(E_tot-E_0)/E_0: "<< (E1-energy0)/energy0<<"\t";
            std::cout << "Accuracy: "<< 2.*(diff-diss)/(diff+diss)<<" d E/dt = " << diff <<" Lambda =" << diss << "\n";
            E0 = E1;

            try{ karniadakis( feltor, rolkar, y0);}
            catch( dg::Fail& fail) { 
                std::cerr << "CG failed to converge to "<<fail.epsilon()<<"\n";
                std::cerr << "Does Simulation respect CFL condition?\n";
                glfwSetWindowShouldClose( w, GL_TRUE);
                break;
            }
        }
        time += (double)p.itstp*p.dt;
#ifdef DG_BENCHMARK
        t.toc();
        std::cout << "\n\t Step "<<step;
        std::cout << "\n\t Average time for one step: "<<t.diff()/(double)p.itstp<<"s\n\n";
#endif//DG_BENCHMARK
    }
    glfwTerminate();
    ////////////////////////////////////////////////////////////////////

    return 0;

}
