#include "hip/hip_runtime.h"
#include <iostream>
#include <iomanip>
#include <vector>
#include <sstream>
#include <cmath>
// #define DG_DEBUG



#include "dg/backend/timer.cuh"
#include "dg/backend/xspacelib.cuh"
#include "dg/backend/interpolation.cuh"
#include "file/read_input.h"
#include "file/nc_utilities.h"
#include "solovev/geometry.h"

#include "feltor.cuh"
#include "parameters.h"

/*
   - reads parameters from input.txt or any other given file, 
   - integrates the ToeflR - functor and 
   - writes outputs to a given outputfile using hdf5. 
        density fields are the real densities in XSPACE ( not logarithmic values)
*/

const unsigned k = 3;//!< a change in k needs a recompilation

int main( int argc, char* argv[])
{
    ////////////////////////Parameter initialisation//////////////////////////
    std::vector<double> v,v3;
    std::string input, geom;
    if( argc != 4)
    {
        std::cerr << "ERROR: Wrong number of arguments!\nUsage: "<< argv[0]<<" [inputfile] [geomfile] [outputfile]\n";
        return -1;
    }
    else 
    {

        try{
            input = file::read_file( argv[1]);
            geom = file::read_file( argv[2]);
            v = file::read_input( argv[1]);
            v3 = file::read_input( argv[2]); 
        }catch( toefl::Message& m){
            m.display();
            std::cout << input << std::endl;
            std::cout << geom << std::endl;
            return -1;
        }
    }
    const eule::Parameters p( v);
    p.display( std::cout);
    const solovev::GeomParameters gp(v3);
    gp.display( std::cout);
    ////////////////////////////////set up computations///////////////////////////

    double Rmin=gp.R_0-p.boxscale*gp.a;
    double Zmin=-p.boxscale*gp.a*gp.elongation;
    double Rmax=gp.R_0+p.boxscale*gp.a; 
    double Zmax=p.boxscale*gp.a*gp.elongation;
    //Make grids
    dg::Grid3d<double > grid( Rmin,Rmax, Zmin,Zmax, 0, 2.*M_PI, p.n, p.Nx, p.Ny, p.Nz, dg::DIR, dg::DIR, dg::PER, dg::cylindrical);  
    dg::Grid3d<double > grid_out( Rmin,Rmax, Zmin,Zmax, 0, 2.*M_PI, p.n_out, p.Nx_out, p.Ny_out, p.Nz_out, dg::DIR, dg::DIR, dg::PER, dg::cylindrical);  
     
    //create RHS 
    std::cout << "Constructing Feltor...\n";
    eule::Feltor<dg::DMatrix, dg::DVec, dg::DVec > feltor( grid, p,gp); 
    std::cout << "Constructing Rolkar...\n";
    eule::Rolkar<dg::DMatrix, dg::DVec, dg::DVec > rolkar( grid, p,gp);
    std::cout << "Done!\n";

    /////////////////////The initial field///////////////////////////////////////////
    //initial perturbation
    //dg::Gaussian3d init0(gp.R_0+p.posX*gp.a, p.posY*gp.a, M_PI, p.sigma, p.sigma, p.sigma, p.amp);
     dg::Gaussian init0( gp.R_0+p.posX*gp.a, p.posY*gp.a, p.sigma, p.sigma, p.amp);

//    dg::BathRZ init0(16,16,p.Nz,Rmin,Zmin, 30.,5.,p.amp);
//     solovev::ZonalFlow init0(p, gp);
    
    //background profile
    solovev::Nprofile prof(p, gp); //initial background profile
    std::vector<dg::DVec> y0(4, dg::evaluate( prof, grid)), y1(y0); 
    
    //field aligning
    //dg::CONSTANT gaussianZ( 1.);
    dg::GaussianZ gaussianZ( M_PI, p.sigma_z*M_PI, 1);
    y1[1] = feltor.dz().evaluate( init0, gaussianZ, (unsigned)p.Nz/2, 3); //rounds =3 ->3*2-1
    y1[2] = dg::evaluate( gaussianZ, grid);
    dg::blas1::pointwiseDot( y1[1], y1[2], y1[1]);
    //no field aligning
    //y1[1] = dg::evaluate( init0, grid);
    
    dg::blas1::axpby( 1., y1[1], 1., y0[1]); //initialize ni
    dg::blas1::transform(y0[1], y0[1], dg::PLUS<>(-1)); //initialize ni-1
    dg::blas1::pointwiseDot(rolkar.damping(),y0[1], y0[1]); //damp with gaussprofdamp
    feltor.initializene( y0[1], y0[0]);    
    dg::blas1::axpby( 0., y0[2], 0., y0[2]); //set Ue = 0
    dg::blas1::axpby( 0., y0[3], 0., y0[3]); //set Ui = 0
    
    dg::Karniadakis< std::vector<dg::DVec> > karniadakis( y0, y0[0].size(), p.eps_time);
    karniadakis.init( feltor, rolkar, y0, p.dt);
    karniadakis( feltor, rolkar, y0); //now energies and potential are at time 0
    /////////////////////////////set up netcdf/////////////////////////////////////
    file::NC_Error_Handle err;
    int ncid;
    err = nc_create( argv[3],NC_NETCDF4|NC_CLOBBER, &ncid);
    err = nc_put_att_text( ncid, NC_GLOBAL, "inputfile", input.size(), input.data());
    err = nc_put_att_text( ncid, NC_GLOBAL, "geomfile", geom.size(), geom.data());
    int dim_ids[4], tvarID;
    err = file::define_dimensions( ncid, dim_ids, &tvarID, grid_out);
    solovev::FieldR fieldR(gp);
    solovev::FieldZ fieldZ(gp);
    solovev::FieldP fieldP(gp);
    dg::HVec vecR = dg::evaluate( fieldR, grid_out);
    dg::HVec vecZ = dg::evaluate( fieldZ, grid_out);
    dg::HVec vecP = dg::evaluate( fieldP, grid_out);
    int vecID[3];
    err = nc_def_var( ncid, "BR", NC_DOUBLE, 3, &dim_ids[1], &vecID[0]);
    err = nc_def_var( ncid, "BZ", NC_DOUBLE, 3, &dim_ids[1], &vecID[1]);
    err = nc_def_var( ncid, "BP", NC_DOUBLE, 3, &dim_ids[1], &vecID[2]);
    err = nc_enddef( ncid);
    err = nc_put_var_double( ncid, vecID[0], vecR.data());
    err = nc_put_var_double( ncid, vecID[1], vecZ.data());
    err = nc_put_var_double( ncid, vecID[2], vecP.data());
    err = nc_redef(ncid);

    //field IDs
    std::string names[5] = {"electrons", "ions", "Ue", "Ui", "potential"}; 
    int dataIDs[5]; 
    for( unsigned i=0; i<5; i++){
        err = nc_def_var( ncid, names[i].data(), NC_DOUBLE, 4, dim_ids, &dataIDs[i]);}

    //energy IDs
    int EtimeID, EtimevarID;
    err = file::define_time( ncid, "energy_time", &EtimeID, &EtimevarID);
    int energyID, massID, energyIDs[5], dissID, dEdtID, accuracyID;
    err = nc_def_var( ncid, "energy",   NC_DOUBLE, 1, &EtimeID, &energyID);
    err = nc_def_var( ncid, "mass",   NC_DOUBLE, 1, &EtimeID, &massID);
    std::string energies[5] = {"Se", "Si", "Uperp", "Upare", "Upari"}; 
    for( unsigned i=0; i<5; i++){
        err = nc_def_var( ncid, energies[i].data(), NC_DOUBLE, 1, &EtimeID, &energyIDs[i]);}
    err = nc_def_var( ncid, "dissipation",   NC_DOUBLE, 1, &EtimeID, &dissID);
    err = nc_def_var( ncid, "dEdt",     NC_DOUBLE, 1, &EtimeID, &dEdtID);
    err = nc_def_var( ncid, "accuracy", NC_DOUBLE, 1, &EtimeID, &accuracyID);
    err = nc_enddef(ncid);
    ///////////////////////////////////first output/////////////////////////
    std::cout << "First output ... \n";
    size_t start[4] = {0, 0, 0, 0};
    size_t count[4] = {1., grid_out.Nz(), grid_out.n()*grid_out.Ny(), grid_out.n()*grid_out.Nx()};
    dg::DVec transfer(  dg::evaluate(dg::zero, grid));
    dg::DVec transferD( dg::evaluate(dg::zero, grid_out));
    dg::HVec transferH( dg::evaluate(dg::zero, grid_out));
    dg::DMatrix interpolate = dg::create::interpolation( grid_out, grid); 
    for( unsigned i=0; i<4; i++)
    {
        dg::blas2::symv( interpolate, karniadakis.last()[i], transferD);
        transferH = transferD;//transfer to host
        err = nc_put_vara_double( ncid, dataIDs[i], start, count, transferH.data() );
    }
    transfer = feltor.potential()[0];
    dg::blas2::symv( interpolate, transfer, transferD);
    transferH = transferD;//transfer to host
    err = nc_put_vara_double( ncid, dataIDs[4], start, count, transferH.data() );
    double time = 0;
    err = nc_put_vara_double( ncid, tvarID, start, count, &time);
    err = nc_put_vara_double( ncid, EtimevarID, start, count, &time);

    size_t Estart[] = {0};
    size_t Ecount[] = {1};
    double energy0 = feltor.energy(), mass0 = feltor.mass(), E0 = energy0, mass = mass0, E1 = 0.0, dEdt = 0., diss = 0., accuracy=0.;
    std::vector<double> evec = feltor.energy_vector();
    err = nc_put_vara_double( ncid, energyID, Estart, Ecount, &energy0);
    err = nc_put_vara_double( ncid, massID,   Estart, Ecount, &mass0);
    for( unsigned i=0; i<5; i++)
        err = nc_put_vara_double( ncid, energyIDs[i], Estart, Ecount, &evec[i]);

    err = nc_put_vara_double( ncid, dissID,     Estart, Ecount,&diss);
    err = nc_put_vara_double( ncid, dEdtID,     Estart, Ecount,&dEdt);
    err = nc_put_vara_double( ncid, accuracyID, Estart, Ecount,&accuracy);
    err = nc_close(ncid);
    std::cout << "First write successful!\n";

    ///////////////////////////////////////Timeloop/////////////////////////////////
    dg::Timer t;
    t.tic();
#ifdef DG_BENCHMARK
    unsigned step = 0;
#endif //DG_BENCHMARK
    for( unsigned i=1; i<=p.maxout; i++)
    {

#ifdef DG_BENCHMARK
        dg::Timer ti;
        ti.tic();
#endif//DG_BENCHMARK
        for( unsigned j=0; j<p.itstp; j++)
        {
            try{ karniadakis( feltor, rolkar, y0);}
            catch( dg::Fail& fail) { 
                std::cerr << "CG failed to converge to "<<fail.epsilon()<<"\n";
                std::cerr << "Does Simulation respect CFL condition?\n";
                err = nc_close(ncid);
                return -1;
            }
            step++;
            time+=p.dt;
            Estart[0] = step;
            E1 = feltor.energy(), mass = feltor.mass(), diss = feltor.energy_diffusion();
            dEdt = (E1 - E0)/p.dt; 
            E0 = E1;
            accuracy = 2.*fabs( (dEdt-diss)/(dEdt + diss));
            evec = feltor.energy_vector();
            err = nc_open(argv[3], NC_WRITE, &ncid);
            err = nc_put_vara_double( ncid, EtimevarID, Estart, Ecount, &time);
            err = nc_put_vara_double( ncid, energyID, Estart, Ecount, &E1);
            err = nc_put_vara_double( ncid, massID,   Estart, Ecount, &mass);
            for( unsigned i=0; i<5; i++)
            {
                err = nc_put_vara_double( ncid, energyIDs[i], Estart, Ecount, &evec[i]);
            }
            err = nc_put_vara_double( ncid, dissID,     Estart, Ecount,&diss);
            err = nc_put_vara_double( ncid, dEdtID,     Estart, Ecount,&dEdt);
            err = nc_put_vara_double( ncid, accuracyID, Estart, Ecount,&accuracy);
            std::cout << "(m_tot-m_0)/m_0: "<< (feltor.mass()-mass0)/mass0<<"\t";
            std::cout << "(E_tot-E_0)/E_0: "<< (E1-energy0)/energy0<<"\t";
            std::cout <<" d E/dt = " << dEdt <<" Lambda = " << diss << " -> Accuracy: "<< accuracy << "\n";
            err = nc_close(ncid);

        }
#ifdef DG_BENCHMARK
        ti.toc();
        std::cout << "\n\t Step "<<step <<" of "<<p.itstp*p.maxout <<" at time "<<time;
        std::cout << "\n\t Average time for one step: "<<ti.diff()/(double)p.itstp<<"s\n\n"<<std::flush;
#endif//DG_BENCHMARK
        //////////////////////////write fields////////////////////////
        start[0] = i;
        err = nc_open(argv[3], NC_WRITE, &ncid);
        for( unsigned j=0; j<4; j++)
        {
            dg::blas2::symv( interpolate, karniadakis.last()[j], transferD);
            transferH = transferD;//transfer to host
            err = nc_put_vara_double( ncid, dataIDs[j], start, count, transferH.data());
        }
        transfer = feltor.potential()[0];
        dg::blas2::symv( interpolate, transfer, transferD);
        transferH = transferD;//transfer to host
        err = nc_put_vara_double( ncid, dataIDs[4], start, count, transferH.data() );
        err = nc_put_vara_double( ncid, tvarID, start, count, &time);
        err = nc_close(ncid);
    }
    t.toc(); 
    unsigned hour = (unsigned)floor(t.diff()/3600);
    unsigned minute = (unsigned)floor( (t.diff() - hour*3600)/60);
    double second = t.diff() - hour*3600 - minute*60;
    std::cout << std::fixed << std::setprecision(2) <<std::setfill('0');
    std::cout <<"Computation Time \t"<<hour<<":"<<std::setw(2)<<minute<<":"<<second<<"\n";
    std::cout <<"which is         \t"<<t.diff()/p.itstp/p.maxout<<"s/step\n";

    return 0;

}

