#include "hip/hip_runtime.h"
#include <iostream>
#include <iomanip>
#include <vector>
#include <fstream>
#include <sstream>
#include <cmath>

#include "dg/backend/xspacelib.cuh"
#include "dg/functors.h"
#include "file/read_input.h"
#include "draw/host_window.h"

#include "geometry.h"
#include "init.h"

int main( int argc, char* argv[])
{
    //std::cout << "Type n, Nx, Ny\n";
    unsigned n, Nx, Ny;
    //std::cin >> n>> Nx>>Ny;   
    std::vector<double> v, v2;

    try{ 
        if( argc==1)
        {
            v = file::read_input( "../feltor/input.txt");
            v2 = file::read_input( "geometry_params.txt"); 
        }
        else
        {
            v = file::read_input( argv[1]); 
            v2 = file::read_input( argv[2]);
        }
    }
    catch (toefl::Message& m) {  
        m.display(); 
        for( unsigned i = 0; i<v.size(); i++)
            std::cout << v[i] << " ";
            std::cout << std::endl;
        return -1;}
    //write parameters from file into variables
    const eule::Parameters p(v);
    const solovev::GeomParameters gp(v2);
    p.display( std::cout);
    gp.display( std::cout);
    n = p.n, Nx = p.Nx, Ny = p.Ny;
    double Rmin=gp.R_0-p.boxscale*gp.a;
    double Zmin=-p.boxscale*gp.a*gp.elongation;
    double Rmax=gp.R_0+p.boxscale*gp.a; 
    double Zmax=p.boxscale*gp.a*gp.elongation;

 
    //construct all geometry quantities
    solovev::Psip psip(gp);
    solovev::PsipR psipR(gp);
    solovev::PsipRR psipRR(gp);  
    solovev::PsipZ psipZ(gp);  
    solovev::PsipZZ psipZZ(gp);   
    solovev::PsipRZ psipRZ(gp);  
    solovev::Ipol ipol(gp);
    solovev::InvB invB(gp);
    solovev::BR bR(gp);
    solovev::BZ bZ(gp);

    //Feltor quantities
    solovev::CurvatureR curvatureR(gp);
    solovev::CurvatureZ curvatureZ(gp);
    solovev::GradLnB gradLnB(gp);
    solovev::Field field(gp);
    solovev::FieldR fieldR(gp);
    solovev::FieldZ fieldZ(gp);
    solovev::Iris iris(gp);
    solovev::Pupil pupil(gp);
    solovev::GaussianDamping dampgauss(gp);
    solovev::GaussianProfDamping dampprof(gp);
    solovev::ZonalFlow zonalflow(p, gp);
    solovev::PsiLimiter psilimiter(gp);
    solovev::Nprofile prof(p, gp);

    dg::BathRZ bath(16,16,p.Nz,Rmin,Zmin, 30.,5.,p.amp);
//     dg::Gaussian3d bath(gp.R_0+p.posX*gp.a, p.posY*gp.a, M_PI, p.sigma, p.sigma, p.sigma, p.amp);
    dg::Gaussian3d blob(gp.R_0+p.posX*gp.a, p.posY*gp.a, M_PI, p.sigma, p.sigma, p.sigma, p.amp);
    dg::Grid2d<double> grid(Rmin,Rmax,Zmin,Zmax, n,Nx,Ny,dg::PER,dg::PER);

    std::vector<dg::HVec> hvisual(21);
        //allocate mem for visual
    std::vector<dg::HVec> visual(21);

    //B field functions
    hvisual[1] = dg::evaluate( psip, grid);
    hvisual[2] = dg::evaluate( ipol, grid);
    hvisual[3] = dg::evaluate( invB, grid);
    hvisual[4] = dg::evaluate( field, grid);
    hvisual[5] = dg::evaluate( curvatureR, grid);
    hvisual[6] = dg::evaluate( curvatureZ, grid);
    hvisual[7] = dg::evaluate( gradLnB, grid);
    //cut functions
    hvisual[8] = dg::evaluate( iris, grid);
    hvisual[9] = dg::evaluate( pupil, grid);
    hvisual[10] = dg::evaluate( dampprof, grid);
    hvisual[11] = dg::evaluate( dampgauss, grid);
    hvisual[12] = dg::evaluate( psilimiter, grid);
    //initial functions
    hvisual[13] = dg::evaluate( zonalflow, grid);
    hvisual[14] = dg::evaluate( prof, grid);
    hvisual[15] = dg::evaluate( blob, grid);
    hvisual[16] = dg::evaluate( bath, grid);

    //initial functions damped and with profile
    hvisual[17] = dg::evaluate( dg::one, grid);
    hvisual[18] = dg::evaluate( dg::one, grid);
    hvisual[19] = dg::evaluate( dg::one, grid);
    hvisual[20] = dg::evaluate( dg::one, grid);            
    dg::blas1::axpby( 1.,hvisual[16] , 1.,hvisual[14],hvisual[17]); //prof + bath
    dg::blas1::axpby( 1.,hvisual[13] , 1.,hvisual[14],hvisual[18]); //prof + zonal
    dg::blas1::axpby( 1.,hvisual[15] , 1.,hvisual[14],hvisual[19]); //prof + blob
    dg::blas1::transform(hvisual[17], hvisual[17], dg::PLUS<>(-1)); //to n -1
    dg::blas1::transform(hvisual[18], hvisual[18], dg::PLUS<>(-1)); //to n -1
    dg::blas1::transform(hvisual[19], hvisual[19], dg::PLUS<>(-1)); //to n -1
    dg::blas1::pointwiseDot(hvisual[10], hvisual[17], hvisual[17]); //damped 
    dg::blas1::pointwiseDot(hvisual[10], hvisual[18], hvisual[18]); //damped 
    dg::blas1::pointwiseDot(hvisual[10], hvisual[19], hvisual[19]); //damped 



//         Compute flux average
    solovev::Alpha alpha(gp); // = B^phi / |nabla psip |
    
    std::cout << "Compute flux average of psi   "<< "\n";
    dg::DVec psipog2d   = dg::evaluate( psip, grid);
    dg::DVec alphaog2d   = dg::evaluate( alpha, grid); 
    
    double psipmin = (float)thrust::reduce( psipog2d .begin(), psipog2d .end(), 0.0,thrust::minimum<double>()  );
    unsigned Npsi = 100;//set number of psivalues
    dg::Grid1d<double> g1d(psipmin ,0.0, 1,Npsi,dg::DIR);
    
    solovev::FluxSurfaceAverage<dg::HVec> fsa1(grid,gp,psipog2d );
    solovev::SafetyFactor<dg::HVec> qprof(grid,gp,alphaog2d );
    dg::HVec fsaofpsip = dg::evaluate(fsa1,g1d);
    dg::HVec sf = dg::evaluate(qprof,g1d);
    dg::HVec abs = dg::evaluate( dg::coo1, g1d);

    
for (unsigned i=0;i<g1d.size() ;i++) {
    std::cout << "psip_ref = " << abs[i] << "  psip_fsa = " << fsaofpsip[i]<< " rel error = " << ( fsaofpsip[i]-abs[i])/abs[i] << "  q = " << sf[i]<<"\n";
}
    
    //make equidistant grid from dggrid
    dg::HMatrix equigrid = dg::create::backscatter(grid);               

    //evaluate on valzues from devicevector on equidistant visual hvisual vector
    for( unsigned i=1; i<=20; i++){
        visual[i] = dg::evaluate( dg::one, grid);
        dg::blas2::gemv( equigrid, hvisual[i], visual[i]);
    }
    //Create Window and set window title
    GLFWwindow* w = draw::glfwInitAndCreateWindow( 1500, 1200, "");
    draw::RenderHostData render(4 , 5);
  
    //create a colormap
    draw::ColorMapRedBlueExtMinMax colors(-1.0, 1.0);

    std::string names[] = { "", "psip", "ipol", "1/B","invbf", "K^R", "K_Z", "gradLnB", 
        "iris", "pupil", "dampprof", "damp", "lim",  
        "zonal", "prof", "blob", "bath", "ini1","ini2","ini3","ini3"};


    std::stringstream title;
    title << std::setprecision(2) << std::scientific;
    while (!glfwWindowShouldClose( w ))
    {
        for(unsigned i=1; i<=20; i++)
        {

            colors.scalemax() = (float)thrust::reduce( visual[i].begin(), visual[i].end(), -100., thrust::maximum<double>()   );
            colors.scalemin() =  (float)thrust::reduce( visual[i].begin(), visual[i].end(), colors.scalemax() ,thrust::minimum<double>() );
//             if(i==1) colors.scalemax() = - colors.scalemin();
//             if(i<=6 && i>=4) colors.scalemax() = - colors.scalemin();
//             if(i==18) colors.scalemin() = 1.0;
            title <<names[i]<<" / "<<colors.scalemin()<<"  " << colors.scalemax()<<"\t";
            render.renderQuad( visual[i], grid.n()*grid.Nx(), grid.n()*grid.Ny(), colors);

        }

        title << std::fixed; 
        glfwSetWindowTitle(w,title.str().c_str());
        title.str("");
        glfwSwapBuffers(w);
        glfwWaitEvents();
    }

    glfwTerminate();
    return 0;
}
