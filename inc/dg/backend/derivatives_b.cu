#include "hip/hip_runtime.h"
#include <iostream>

#include "blas.h"
#include "derivatives.cuh"
#include "evaluation.cuh"
#include "typedefs.cuh"

#include "timer.cuh"

const double lx = 2*M_PI;
/*
double function( double x, double y, double z) { return sin(3./4.*z);}
double derivative( double x, double y, double z) { return 3./4.*cos(3./4.*z);}
dg::bc bcz = dg::DIR_NEU;
*/
double function  ( double x, double y) { return sin(x);}
double derivative( double x, double y) { return cos(x);}
dg::bc bcx = dg::DIR;

void multiply( dg::Operator<double>& op, const double* x, double* y)
{
    for( unsigned i=0; i<op.size(); i++)
    {
        y[i] = 0;
        for( unsigned j=0; j<op.size(); j++)
            y[i]+= op(i,j)*x[j];
    }
}

int main()
{
    dg::Timer t;
    unsigned n, Nx, Ny;
    std::cout << "Note the supraconvergence!\n";
    std::cout << "Type in n, Nx and Ny!\n";
    std::cin >> n >> Nx >> Ny;
    std::cout << "# of cells          " << Nx*Ny <<"\n";
    std::cout << "# of polynomials    " << n <<"\n";
    dg::Grid2d<double> g( 0, lx, 0, lx, n, Nx, Ny, bcx, dg::PER);
    dg::DMatrix dx = dg::create::dx( g, bcx);
    dg::DMatrix lxM = dg::create::laplacianM( g, bcx, dg::PER, dg::normed, dg::centered);
    const dg::DVec hv = dg::evaluate( function, g);
    dg::DVec hw = hv;
    const dg::DVec hu = dg::evaluate( derivative, g);


    t.tic();
    for( unsigned i=0; i<10; i++)
        dg::blas2::symv( dx, hv, hw);
    t.toc();
    std::cout << "Evaluation of dx took "<<t.diff()/10.<<"s\n";
    dg::blas1::axpby( 1., hu, -1., hw);
    std::cout << "Distance to true solution: "<<sqrt(dg::blas2::dot(hw, (dg::DVec)dg::create::weights(g), hw))<<"\n";
    t.tic();
    for( unsigned i=0; i<10; i++)
        dg::blas2::symv( lxM, hv, hw);
    t.toc();
    std::cout << "Evaluation of Lx took "<<t.diff()/10.<<"s\n";
    dg::blas1::axpby( 1., hv, -1., hw);
    std::cout << "Distance to true solution: "<<sqrt(dg::blas2::dot(hw, (dg::DVec)dg::create::weights(g), hw))<<"\n";
    //for periodic bc | dirichlet bc
    //n = 1 -> p = 2      2
    //n = 2 -> p = 1      1
    //n = 3 -> p = 3      3
    //n = 4 -> p = 3      3
    //n = 5 -> p = 5      5
    std::cout << "TEST VARIOUS HOST VERSIONS OF DERIVATIVE!\n";
    dg::Operator<double> forw( g.dlt().forward( ));
    dg::Operator<double> back( g.dlt().forward( ));
    dg::HMatrix DX(dx);
    const dg::HVec v = dg::evaluate( function, g);
    dg::HVec w(v);
    t.tic();
    double temp[n];
    for( unsigned i=0; i<10; i++)
        for( unsigned k=0; k<(Ny-1)*(Nx-1); k++)
        {
            multiply( forw, &v[k*n], &w[k*n]);
            multiply( back, &v[(k+1)*n], temp);
            for( unsigned j=0; j<n; j++)
                w[k*n+j] += temp[j];
        }
    t.toc();
    std::cout << "Evaluation of host derivative took "<<t.diff()/10.<<"s\n";
    t.tic();
    for( unsigned i=0; i<10; i++)
        dg::blas2::symv( DX, v, w);
    t.toc();
    std::cout << "Evaluation of host derivative took "<<t.diff()/10.<<"s\n";

    return 0;
}
