#include "hip/hip_runtime.h"
#include <iostream>
#include <cmath>

#include <thrust/host_vector.h>
#include <thrust/device_vector.h>

#include "evaluation.cuh"
#include "weights.cuh"

#include "blas.h"

double function( double x)
{
    return exp(x);
}

double function( double x, double y)
{
        return exp(x)*exp(y);
}
double function( double x, double y, double z)
{
        return exp(x)*exp(y)*exp(z);
}

const double lx = 2;
const double ly = 2;
const double lz = 2;

typedef thrust::device_vector< double>   DVec;
typedef thrust::host_vector< double>     HVec;

using namespace std;
int main()
{
    unsigned n;
    cout << "Type # of polynomial coefficients ( 1, 2,...,20)!\n";
    cin >> n;
    unsigned N, Nx, Ny, Nz;
    cout << "# of polynomial coefficients is: "<< n<<endl;
    cout << "Type # of grid cells (e.g. 10, 100)! ( Nx = N, Ny = 2N, Nz = 10*N)\n";
    cin >> N;
    cout << "# of grid cells is: "<< N<<endl;
    Nx = N; Ny = 2*N; Nz = 10*N;

    dg::Grid1d<double> g1d( 0, lx, n, N);
    dg::Grid2d<double> g2d( 0, lx,0, ly,n, Nx, Ny);
    dg::Grid3d<double> g3d( 0, lx,0, ly,0, lz, n, Nx, Ny, Nz,dg::PER,dg::PER,dg::PER,dg::cylindrical);

    //test evaluation functions
    HVec h_x = dg::evaluate( function, g1d);
    HVec h_n = dg::evaluate( function, g2d);
    HVec h_z = dg::evaluate( function, g3d);
    HVec w3d = dg::create::weights( g3d);

    //test preconditioners
    double normX = dg::blas2::dot( h_x, dg::create::weights(g1d), h_x);
    double norm2X = dg::blas2::dot( dg::create::weights(g2d), h_n);
    double norm3X = dg::blas2::dot( h_z, w3d, h_z);

    cout << "Square normalized 1DXnorm "<< normX <<"\n";
    double solution = (exp(4.) -exp(0))/2.;
    cout << "Correct square norm is    "<<solution<<endl;
    cout << "Square normalized 2DXnorm "<< norm2X<<"\n";
    double solution2 = (exp(4.)-exp(0))/2.*(exp(4.) -exp(0))/2.;
    cout << "Correct square norm is    "<<solution2<<endl;

    cout << "Square normalized 3DXnorm   "<< norm3X<<"\n";
    if( g3d.system() == dg::cylindrical)
        solution = (3*exp(4.)+1.)/4.;
    double solution3 = solution2*solution;
    cout << "Correct square norm is      "<<solution3<<endl;
    cout << "Relative 3d error is        "<<(norm3X-solution3)/solution3<<"\n";
    int keys[10] = {0,0,0,1,1,2,0,0,0,0};
    int vals[10] = {1,1,1,1,1,1,1,1,1,1};
    thrust::exclusive_scan_by_key( keys, keys+10, vals, vals);
    for(unsigned i=0; i<10; i++)
        std::cout << vals[i]<<" ";
    std::cout << std::endl;
    int gatherIdx[10] = {0,1,0,1,0,1,2,3,2,3};
    double input[4] = {0,1,2,3};
    double output[10];
    thrust::gather( gatherIdx, gatherIdx+10, input, output);
    for(unsigned i=0; i<10; i++)
        std::cout << output[i]<<" ";
    std::cout << std::endl;
    return 0;
} 
