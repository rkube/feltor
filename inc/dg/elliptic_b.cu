#include "hip/hip_runtime.h"
#include <iostream>
#include <iomanip>

#include <thrust/host_vector.h>
#include <thrust/device_vector.h>

#include "backend/timer.cuh"
#include "backend/evaluation.cuh"
#include "backend/derivatives.cuh"
#include "backend/typedefs.cuh"
#include "backend/cusp_thrust_backend.h"

#include "cg.h"
#include "elliptic.h"


const double R_0 = 1000;
const double lx = 2.*M_PI;
const double ly = 2.*M_PI;
const double lz = 2.*M_PI;
double fct(double x, double y, double z){ return sin(x-R_0)*sin(y);}
double derivative( double x, double y, double z){return cos(x-R_0)*sin(y);}
double laplace_fct( double x, double y, double z) { return -1./x*cos(x-R_0)*sin(y) + 2.*sin(y)*sin(x-R_0);}
dg::bc bcx = dg::DIR;
double initial( double x, double y, double z) {return sin(0);}


int main()
{
    dg::Timer t;
    unsigned n, Nx, Ny, Nz; 
    std::cout << "Type n, Nx, Ny and Nz\n";
    std::cin >> n >> Nx >> Ny >> Nz;
    double eps;
    std::cout << "Type epsilon! \n";
    std::cin >> eps;
    dg::Grid3d<double> grid( R_0, R_0+lx, 0, ly, 0,lz, n, Nx, Ny,Nz, bcx, dg::PER, dg::PER, dg::cylindrical);
    dg::DVec w3d = dg::create::weights( grid);
    dg::DVec v3d = dg::create::inv_weights( grid);
    dg::DVec x = dg::evaluate( initial, grid);

    std::cout << "Create Laplacian\n";
    t.tic();
    dg::Elliptic<dg::DMatrix, dg::DVec, dg::DVec> laplace(grid, dg::not_normed, dg::centered);
    dg::DMatrix DX = dg::create::dx( grid);
    t.toc();
    std::cout<< "Creation took "<<t.diff()<<"s\n";

    dg::CG< dg::DVec > pcg( x, n*n*Nx*Ny);

    std::cout<<"Expand right hand side\n";
    const dg::DVec solution = dg::evaluate ( fct, grid);
    const dg::DVec deriv = dg::evaluate( derivative, grid);
    dg::DVec b = dg::evaluate ( laplace_fct, grid);
    //compute W b
    dg::blas2::symv( w3d, b, b);
    
    std::cout << "For a precision of "<< eps<<" ..."<<std::endl;
    t.tic();
    std::cout << "Number of pcg iterations "<< pcg( laplace, x, b, v3d, eps)<<std::endl;
    t.toc();
    std::cout << "... on the device took "<< t.diff()<<"s\n";
    dg::DVec  error(  solution);
    dg::blas1::axpby( 1., x,-1., error);

    double normerr = dg::blas2::dot( w3d, error);
    double norm = dg::blas2::dot( w3d, solution);
    std::cout << "L2 Norm of relative error is:               " <<sqrt( normerr/norm)<<std::endl;
    dg::blas2::gemv( DX, x, error);
    dg::blas1::axpby( 1., deriv, -1., error);
    normerr = dg::blas2::dot( w3d, error); 
    norm = dg::blas2::dot( w3d, deriv);
    std::cout << "L2 Norm of relative error in derivative is: " <<sqrt( normerr/norm)<<std::endl;
    //both function and derivative converge with order P 

    return 0;
}
